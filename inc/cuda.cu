#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cassert>
#include "FixedVectorMath.hh"
#include <cmath>
#include <chrono>
#include <iostream>
#include <cudaProfiler.h>
#include <cstring>
#include <omp.h>

void checkCudaError(const char* message) {
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << message << " - " << hipGetErrorString(err) << std::endl;
        exit(-1);
    }
}
#define CEIL_DIV(M, N) (((M) + (N)-1) / (N))
#define TILE_SIZE 32

/*
    This file is split into two sections!

    1. Kernel code
        - This is the code executed on the GPU itself
    
    2. Cuda Definitions
        - Our API to interact with the GPU, responsible for data preperation, transmission, and return

    ////////////////////////////////////////
    // Important Variables
    ////////////////////////////////////////

    blockIdx: Which block (in a given dimension denoted by .(x,y,z)) the current thread belongs to.

    blockDim: How many threads per block along the given axis .(x, y, z)
        - Nvidia executes threads in groups of 32 called warps. It's optimal (not required) to keep block sizes a multiple of 32


    Refer to the Cheat Sheet for visualizing these dimensions. 
    https://www.eecs.umich.edu/courses/eecs471/resources/materials/CUDA-Thread-Indexing-Cheatsheet.pdf
*/

/*
-----------------------------------------
    Kernel function defintions
-----------------------------------------
*/


void print_attention_scores_per_head(float* d_attention_scores, int num_heads, int sequence_len) {
    float* h_attention_scores = new float[num_heads * sequence_len * sequence_len];
    hipMemcpy(h_attention_scores, d_attention_scores, num_heads * sequence_len * sequence_len * sizeof(float), hipMemcpyDeviceToHost);

    for (int head = 0; head < num_heads; ++head) {
        std::cout << "Head " << head << " Attention Scores:\n";
        for (int i = 0; i < sequence_len; ++i) {
            for (int j = 0; j < sequence_len; ++j) {
                std::cout << h_attention_scores[head * sequence_len * sequence_len + i * sequence_len + j] << " ";
            }
            std::cout << "\n";
        }
        std::cout << "\n";
    }
    delete[] h_attention_scores;
}

void print_device_matrix(const float* d_matrix, int rows, int cols, const std::string& name, hipStream_t stream) {
    // Allocate host memory
    float* h_matrix = new float[rows * cols];

    // Copy from device to host
    hipMemcpyAsync(h_matrix, d_matrix, rows * cols * sizeof(float), hipMemcpyDeviceToHost,stream);
    hipDeviceSynchronize();  // Ensure memory copy is completed before printing

    // Print the first 5x5 submatrix
    std::cout << "First 5x5 submatrix of " << name << ":\n";
    for (int i = 0; i < 24 && i < rows; ++i) {  // Ensure within bounds
        std::cout << i << ":";
        for (int j = 0; j < 70 && j < cols; ++j) {
            std::cout << h_matrix[i * cols + j] << " ";
        }
        std::cout << "\n";
    }

    // Free host memory
    delete[] h_matrix;
}

void printmatrix(const FixedVector<FixedVector<float>>& matrix, const std::string& name) {
    std::cout << "Matrix: " << name << std::endl;
    for (size_t i = 0; i < std::min(matrix.size(), static_cast<size_t>(5)); ++i) {
        for (size_t j = 0; j < std::min(matrix[i].size(), static_cast<size_t>(5)); ++j) {
            std::cout << matrix[i][j] << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
  }
  
void print_dmask(float* dmask, int m) {
    // Allocate host memory
    float* hmask = new float[m * m];

    // Copy from device to host
    hipMemcpy(hmask, dmask, m * m * sizeof(float), hipMemcpyDeviceToHost);

    // Print the first 5x5 submatrix
    std::cout << "First 5x5 submatrix of dmask:\n";
    for (int i = 0; i < 5 && i < m; ++i) {  // Ensure within bounds
        for (int j = 0; j < 5 && j < m; ++j) {
            std::cout << hmask[i * m + j] << " ";
        }
        std::cout << "\n";
    }

    // Free host memory
    delete[] hmask;
}

void print_hmask(const float* hmask, int m) {
    std::cout << "First 5x5 submatrix of hmask:\n";
    for (int i = 0; i < 5 && i < m; ++i) {  // Ensure within bounds
        for (int j = 0; j < 5 && j < m; ++j) {
            std::cout << hmask[i * m + j] << " ";
        }
        std::cout << "\n";
    }
}

void printMatrix(const char* name, float* d_matrix, int rows, int cols) {
    FixedVector<float> h_matrix(rows * cols);
    hipMemcpy(h_matrix.data(), d_matrix, rows * cols * sizeof(float), hipMemcpyDeviceToHost);

    std::cout << name << " (First 5x5 values):\n";
    int display_rows = std::min(5, rows);
    int display_cols = std::min(5, cols);
    
    for (int i = 0; i < display_rows; ++i) {
        for (int j = 0; j < display_cols; ++j) {
            std::cout << h_matrix[i * cols + j] << " ";
        }
        std::cout << "\n";
    }
    std::cout << "--------------------------------------\n";
}


__global__ void mulKernel(const float* A, const float* B, float* out, size_t N){
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N){
        out[idx] = A[idx] * B[idx];
    }
}

__global__ void mul2DKernel(const float* A, const float* B, float* out, size_t width, size_t height){
    //1. Determine row and column for the current thread
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;

    // 2. Check Boundary 
    if (row < height && col < width){
        size_t idx = row * width + col;
        out[idx] = A[idx] * B[idx];
    }
}

__global__ void addKernel(float* A, const float* B, size_t N){
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N){
        A[idx] += B[idx];
    }
}

__global__ void add2DKernel(float* A, const float* B, size_t width, size_t height){
    //1. Determine row and column for the current thread
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;

    // 2. Check Boundary 
    if (row < height && col < width){
        size_t idx = row * width + col;
        A[idx] += B[idx];
    }
}

__global__ void linear_kernel(int M, int N, int K, const float *A, const float *B, const float *bias, float *C) {
    const uint x = blockIdx.x * blockDim.x + threadIdx.x;
    const uint y = blockIdx.y * blockDim.y + threadIdx.y;
    // keep within tile bounds 
    if (x < M && y < N) {
      float tmp = 0.0;
      for (int i = 0; i < K; ++i) 
        tmp += A[x * K + i] * B[i * N + y];
      C[x * N + y] =  tmp + C[x * N + y];
    }
    C[x * N + y] += bias[y];
}

__global__ void sgemm_naive(int M, int N, int K, const float *A,
                            const float *B, float *C) {
  const uint x = blockIdx.x * blockDim.x + threadIdx.x;
  const uint y = blockIdx.y * blockDim.y + threadIdx.y;
  // keep within tile bounds 
  if (x < M && y < N) {
    float tmp = 0.0;
    for (int i = 0; i < K; ++i) 
      tmp += A[x * K + i] * B[i * N + y];
    C[x * N + y] =  tmp + C[x * N + y];
  }
}

__global__ void elementWiseMultiplyKernel(float* dA, float* dB, float* dOut, int m, int n)
{
    // Calculate global thread index
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Perform the element-wise multiplication if within bounds
    if (row < m && col < n)
    {
        int index = row * n + col;
        dOut[index] = dA[index] * dB[index];
    }
}

__global__ void sgemm_blockheads(int m, int n, int k, 
    const float* sequence_history, const float *w_q, const float *w_v, const float *w_k,
    float* Q, float* V, float* K) {

    __shared__ float tile_A[TILE_SIZE][TILE_SIZE];
    __shared__ float tile_Bq[TILE_SIZE][TILE_SIZE];
    __shared__ float tile_Bv[TILE_SIZE][TILE_SIZE];
    __shared__ float tile_Bk[TILE_SIZE][TILE_SIZE];

    int tx = threadIdx.x, ty = threadIdx.y;
    int row = blockIdx.y * TILE_SIZE + ty;
    int col = blockIdx.x * TILE_SIZE + tx;

    float sumQ = 0.0f, sumV = 0.0f, sumK = 0.0f;

    for (int t = 0; t < (k + TILE_SIZE - 1) / TILE_SIZE; ++t) {
        // Load elements into shared memory
        if (row < m && t * TILE_SIZE + tx < k) {
            tile_A[ty][tx] = sequence_history[row * k + t * TILE_SIZE + tx];
        } else {
            tile_A[ty][tx] = 0.0f;
        }

        if (t * TILE_SIZE + ty < k && col < n) {
            tile_Bq[ty][tx] = w_q[(t * TILE_SIZE + ty) * n + col];
            tile_Bv[ty][tx] = w_v[(t * TILE_SIZE + ty) * n + col];
            tile_Bk[ty][tx] = w_k[(t * TILE_SIZE + ty) * n + col];
        } else {
            tile_Bq[ty][tx] = 0.0f;
            tile_Bv[ty][tx] = 0.0f;
            tile_Bk[ty][tx] = 0.0f;
        }

        __syncthreads();

        // Compute partial sum within tile
        for (int i = 0; i < TILE_SIZE; ++i) {
            sumQ += tile_A[ty][i] * tile_Bq[i][tx];
            sumV += tile_A[ty][i] * tile_Bv[i][tx];
            sumK += tile_A[ty][i] * tile_Bk[i][tx];
        }
        __syncthreads();
    }

    if (row < m && col < n) {
        Q[row * n + col] += sumQ;
        V[row * n + col] += sumV;
        K[row * n + col] += sumK;
    }
}

__global__ void compute_attention_scores(
    const float* Q, const float* K, const float* mask,
    float* attention_scores, int sequence_len, int d_head,
    bool use_mask, int num_heads, int d_model) {

    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    int head = blockIdx.z;  // Each block in the z-dimension corresponds to a different head

    if (row >= sequence_len || col >= sequence_len || head >= num_heads) return;

    int head_offset = head * d_head;
    float score = 0.0f;

    // Compute raw attention scores (QK^T)
    for (int k = 0; k < d_head; ++k) {
        int Q_index = row * d_model + head_offset + k;
        int K_index = col * d_model + head_offset + k;
        score += Q[Q_index] * K[K_index];
    }

    // Scale the score
    score /= sqrtf(static_cast<float>(d_head));

    // Apply mask if necessary
    if (use_mask) {
        int mask_index = row * sequence_len + col;
        score += mask[mask_index];  // Ensure mask is correctly indexed
    }

    // Store attention score for this head
    int output_index = head * sequence_len * sequence_len + row * sequence_len + col;
    attention_scores[output_index] = score;
}


__global__ void compute_head_out(
    float* output, const float* att_score_soft, const float* V, 
    const int sequence_len, const int d_model, const int head, const int d_head) 
{
    const int row = blockIdx.x * blockDim.x + threadIdx.x;
    const int col = blockIdx.y * blockDim.y + threadIdx.y;
    
    int head_offset = head * d_head;

    if (row < sequence_len && col < d_head) {
        float tmp = 0.0f;
        for (int i = 0; i < sequence_len; ++i) {
            tmp += att_score_soft[row * sequence_len + i] * V[i * d_model + head_offset + col];
        }
        output[row * d_head + col] = tmp;
    }
}


__global__ void merge_heads(const float* head_out,  float* attention_out, int sequence_len, int d_model, int num_heads, int d_head, int head) // Head index passed as input
{
    int seq_idx = blockIdx.x * blockDim.x + threadIdx.x; // Row index (sequence)
    int feature_idx = blockIdx.y * blockDim.y + threadIdx.y; // Column index (within d_head)

    if (seq_idx < sequence_len && feature_idx < d_head) {
        // Compute correct input index in head_out for this head
        int head_out_index = seq_idx * d_head + feature_idx;

        // Compute correct output index in attention_out
        int attention_out_index = seq_idx * d_model + head * d_head + feature_idx;
        __syncthreads();
        attention_out[attention_out_index] = head_out[head_out_index];
    }
}


__global__ void softMax(float* output, float* input, int M, int N) 
{
    int row = blockDim.x * blockIdx.x + threadIdx.x;

    if (row < M) {
        // maximum of this row
        float x_max = -INFINITY;
        // norm factor of this row
        float norm = 0.0f;

        // output in 3 passes
        for (int col = 0; col < N; col++) {
            int i = row * N + col;
            x_max = max(x_max, input[i]);
        }
        for (int col = 0; col < N; col++) {
            int i = row * N + col;
            norm += expf(input[i] - x_max);
        }
        for (int col = 0; col < N; col++) {
            int i = row * N + col;
            output[i] = expf(input[i] - x_max) / norm;
        }
    }
}

#define BLOCK_SIZE 32
/*
-----------------------------------------
    Cuda API Definitions
-----------------------------------------
*/   
namespace FixedVectorMath {

    FixedVector<FixedVector<float>> MMA_CUDA(
        bool use_mask,
        int sequence_len,
        int d_model,
        const int num_ma_heads, 
        FixedVector<FixedVector<float>> &sequence_history,
        FixedVector<FixedVector<float>> &w_q,
        FixedVector<FixedVector<float>> &w_k,
        FixedVector<FixedVector<float>> &w_v,
        FixedVector<FixedVector<float>> &w_o) {
         
        hipStream_t streams[num_ma_heads];
        for (int i = 0; i < num_ma_heads; i++)
            hipStreamCreate(&streams[i]);
    
        FixedVector<FixedVector<float>> attention_out(sequence_len, FixedVector<float>(d_model, 0.0f));
        FixedVector<FixedVector<float>> mask(sequence_len, FixedVector<float>(sequence_len, 0.0f));
       
        if (use_mask)
          FixedVectorMath::applyMask(mask);
        

        // Fix dimensions
        int m = sequence_len;  // Number of sequences
        int n = d_model;       // Model dimension
        int k = d_model;       // Embedding dimension
        int d_head = d_model / num_ma_heads;
        int num_heads = num_ma_heads;

        // Allocate host memory
        float* hsequence_history = new float[m * k];
        float* hmask = new float[m * m];  // Flattened mask
        float* hout = new float[m * d_model];  
        float* hw_q = new float[k * d_model];
        float* hw_v = new float[k * d_model];
        float* hw_k = new float[k * d_model];
        
        for (size_t i = 0; i < m; ++i) {
            for (size_t j = 0; j < m; ++j) {
                hmask[i * m + j] = mask[i][j];  // Flattening row-major order
            }
        }
       
        // Copy input matrices to 1D arrays
        for (size_t i = 0; i < m; ++i) {
            memcpy(hsequence_history + i * k, sequence_history[i].data(), k * sizeof(float));
        }
        for (size_t i = 0; i < k; ++i) {
            memcpy(hw_q + i * d_model, w_q[i].data(), d_model * sizeof(float));
            memcpy(hw_v + i * d_model, w_v[i].data(), d_model * sizeof(float));
            memcpy(hw_k + i * d_model, w_k[i].data(), d_model * sizeof(float));
        }
    
       // Device memory
        static float* dsequence_history = nullptr;
        static float* dattention_scores = nullptr;
        static float* dattention_scores_softmax = nullptr;
        static float* dOut = nullptr;
        static float* dw_q = nullptr;
        static float* dmask = nullptr;
        static float* dw_k = nullptr;
        static float* dw_v = nullptr;
        static float* dQ = nullptr;
        static float* dV = nullptr;
        static float* dK = nullptr;
        static float* dhead_out = nullptr;
        static float* dattention_out = nullptr;

        if (dsequence_history == nullptr) {
            // Free previously allocated memory (if any)
            hipFree(dw_q);
            hipFree(dw_k);
            hipFree(dw_v);
            hipFree(dmask);
            hipFree(dsequence_history);
            hipFree(dattention_scores);
            hipFree(dQ);
            hipFree(dV);
            hipFree(dOut);
            hipFree(dK);
            hipFree(dhead_out);
            hipFree(dattention_out);

            // Allocate device memory
            hipMalloc((void**)&dsequence_history, m * k * sizeof(float));
            checkCudaError("hipMalloc for dsequence_history");
            
            hipMalloc((void**)&dw_q, k * d_model * sizeof(float));
            checkCudaError("hipMalloc for dw_q");
            
            hipMalloc((void**)&dw_k, k * d_model * sizeof(float));
            checkCudaError("hipMalloc for dw_k");
            
            hipMalloc((void**)&dw_v, k * d_model * sizeof(float));
            checkCudaError("hipMalloc for dw_v");
            
            hipMalloc((void**)&dQ, sequence_len * d_model * sizeof(float));
            checkCudaError("hipMalloc for dQ");
            
            hipMalloc((void**)&dV, sequence_len * d_model * sizeof(float));
            checkCudaError("hipMalloc for dV");
            
            hipMalloc((void**)&dK, sequence_len * d_model * sizeof(float));
            checkCudaError("hipMalloc for dK");
            
            hipMalloc((void**)&dOut, sequence_len * d_model * sizeof(float));
            checkCudaError("hipMalloc for dOut");
            
            hipMalloc((void**)&dattention_scores, num_ma_heads*m * m * sizeof(float));
            checkCudaError("hipMalloc for dattention_scores");

            hipMalloc((void**)&dattention_scores_softmax, m * m * sizeof(float));
            checkCudaError("hipMalloc for dattention_scores");
            
            hipMalloc((void**)&dmask, m * m * sizeof(float));
            checkCudaError("hipMalloc for dmask");
            
            // Allocate memory for head_out (stores attention-weighted V output)
            hipMalloc((void**)&dhead_out, sequence_len * d_head * sizeof(float));
            checkCudaError("hipMalloc for dhead_out");

            // Allocate memory for final attention_out (concatenated head_out results)
            hipMalloc((void**)&dattention_out, sequence_len * d_model * sizeof(float));
            checkCudaError("hipMalloc for dattention_out");
        }

    // Copy host data to device memory
    hipMemcpyAsync(dmask, hmask, m * m * sizeof(float), hipMemcpyHostToDevice,0);
    hipMemcpyAsync(dsequence_history, hsequence_history, m * k * sizeof(float), hipMemcpyHostToDevice,0);
    hipMemcpyAsync(dw_q, hw_q, k * d_model * sizeof(float), hipMemcpyHostToDevice,0);
    hipMemcpyAsync(dw_v, hw_v, k * d_model * sizeof(float), hipMemcpyHostToDevice,0);
    hipMemcpyAsync(dw_k, hw_k, k * d_model * sizeof(float), hipMemcpyHostToDevice,0);

    // Initialize device output matrices
    hipMemsetAsync(dQ, 0, sequence_len * d_model * sizeof(float),0);
    hipMemsetAsync(dV, 0, sequence_len * d_model * sizeof(float),0);
    hipMemsetAsync(dK, 0, sequence_len * d_model * sizeof(float),0);
    hipMemsetAsync(dattention_scores, 0, num_ma_heads*m * m * sizeof(float),0);
    hipMemsetAsync(dattention_scores_softmax, 0, m * m * sizeof(float),0);
    hipMemsetAsync(dhead_out, 0, sequence_len * d_head * sizeof(float),0);
    hipMemsetAsync(dattention_out, 0, sequence_len * d_model * sizeof(float), 0);

    // Kernel configuration
    dim3 blockDim1(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim1((n + BLOCK_SIZE - 1) / BLOCK_SIZE, (m + BLOCK_SIZE - 1) / BLOCK_SIZE);        
    sgemm_blockheads<<<gridDim1, blockDim1, 0, 0>>>(m, n, k, dsequence_history, dw_q, dw_v, dw_k, dQ, dV, dK);
    hipDeviceSynchronize();

    dim3 blockSize(8, 8, 1);  // Adjust based on available resources
    dim3 gridSize(
    (sequence_len + blockSize.x - 1) / blockSize.x,
    (sequence_len + blockSize.y - 1) / blockSize.y,
     num_heads);
    // Each head gets its own block in the z-dimension

    compute_attention_scores<<<gridSize, blockSize>>>(dQ, dK, dmask, dattention_scores, sequence_len, d_head, use_mask, num_ma_heads, d_model);
    print_attention_scores_per_head(dattention_scores, num_heads, sequence_len);
    hipDeviceSynchronize();
    // dim3 blockDim1(16, 16);
    // dim3 gridDim1((d_model + blockDim.x - 1) / blockDim.x, (d_model + blockDim.y - 1) / blockDim.y); 
    // for(int head = 0; head < num_ma_heads; ++head)
    // {
    //     compute_attention_scores<<<gridDim1, blockDim1,0,streams[head]>>>(dQ, dK, dmask, dattention_scores, sequence_len, d_head, use_mask, head, d_model);
    //     softMax<<<gridDim1, blockDim1,0,streams[head]>>>(dattention_scores_softmax,dattention_scores,sequence_len,sequence_len);
    //     compute_head_out<<<gridDim1, blockDim1,0,streams[head]>>>(dhead_out, dattention_scores_softmax,dV, sequence_len,d_model,head, d_head);
    //     hipDeviceSynchronize();
    //     merge_heads<<<gridDim1, blockDim1,0,streams[head]>>>(dhead_out, dattention_out, sequence_len,d_model, num_ma_heads, d_head,head);
    //     // print_device_matrix(dattention_scores, sequence_len, sequence_len, "dattention_out",streams[0]);
    //     // print_device_matrix(dhead_out, sequence_len, d_head, "dhead_out",streams[0]);
    // }
    // print_device_matrix(dattention_out, sequence_len, d_model, "dattention_out",streams[0]);

        // Cleanup
        delete[] hsequence_history;
        delete[] hout;
        delete[] hmask;
        delete[] hw_q;
        delete[] hw_v;
        delete[] hw_k;

        for (int i = 0; i < num_ma_heads; i++)
            hipStreamDestroy(streams[i]);
        
        return attention_out;
    }
    
    
    FixedVector<FixedVector<float>> dotProductCuda(
        FixedVector<FixedVector<float>>& A, 
        FixedVector<FixedVector<float>>& B
        ) {
        
        static hipStream_t stream1;
        // std::cout << stream1;
        if (stream1 == 0) {
            hipStreamCreate (&stream1);
        }
        int m = A.size();  // Number of rows in A
        int n = A[0].size();  // Number of columns in A (also number of rows in B)
        int k = B[0].size();  // Number of columns in B

        // Flatten the 2D vectors into contiguous 1D arrays
        float* hA = new float[m * n];
        float* hB = new float[n * k];
        float* hOut = new float[m * k];

        for (size_t i = 0; i < m; ++i) {
            memcpy(hA + i * n, A[i].data(), n * sizeof(float));
        }
        for (size_t i = 0; i < n; ++i) {
            memcpy(hB + i * k, B[i].data(), k * sizeof(float));
        }

        // Device memory allocations (only if dimensions have changed)
        static float* dA = nullptr;
        static float* dB = nullptr;
        static float* dOut = nullptr;

        static int prevM = -1, prevN = -1, prevK = -1;

        // Check if the previous dimensions were the same, if not we need to 
        // free the pointers on the device
        if (prevM != m || prevK != k || prevN != n) {
            printf("resize");
            hipFreeAsync(dA,stream1);
            hipFreeAsync(dB,stream1);
            hipFreeAsync(dOut,stream1);

            // Allocate device memory
            hipMalloc((void**)&dA, m * n * sizeof(float));
            checkCudaError("hipMalloc for dA");
            hipMalloc((void**)&dB, n * k * sizeof(float));
            checkCudaError("hipMalloc for dB");
            hipMalloc((void**)&dOut, m * k * sizeof(float));
            checkCudaError("hipMalloc for dOut");

            prevM = m;
            prevK = k;
            prevN = n;
        }

        // Copy A and B to device memory asynchronously
        hipMemcpyAsync(dA, hA, m * n * sizeof(float), hipMemcpyHostToDevice,stream1);
        checkCudaError("hipMemcpyAsync for dA");
        hipMemcpyAsync(dB, hB, k * n * sizeof(float), hipMemcpyHostToDevice,stream1);
        checkCudaError("hipMemcpyAsync for dB");
        hipMemsetAsync(dOut, 0, m * k * sizeof(float),stream1);
        checkCudaError("hipMemsetAsync for dOut");

        // Launch the kernel
        dim3 blockDim(16, 16);
        dim3 gridDim((m + blockDim.x - 1) / blockDim.x, (n + blockDim.y - 1) / blockDim.y);  // Grid size based on matrix dimensions
        sgemm_naive<<<gridDim, blockDim,0,stream1>>>(m, n, k, dA, dB, dOut);
        checkCudaError("Kernel launch failed");
        
        
     
        // Copy result back to host asynchronously
       

        //start = std::chrono::high_resolution_clock::now();

        hipMemcpyAsync(hOut, dOut, m * k * sizeof(float), hipMemcpyDeviceToHost,stream1);
        checkCudaError("hipMemcpyAsync for hOut");
        //finish = std::chrono::high_resolution_clock::now();
        //std::cout << std::chrono::duration_cast<std::chrono::nanoseconds>(finish-start).count() << " ns for memcpy3 \n";

        // Convert output back to FixedVector
        
        FixedVector<FixedVector<float>> out(m, FixedVector<float>(k, 0.0f));
        for (size_t i = 0; i < m; ++i) {
            memcpy(out[i].data(), hOut + i * k, k * sizeof(float));  // Copy row i of the result
        }

        
        // Cleanup (free memory)
        delete[] hA;
        delete[] hB;
        delete[] hOut;

        return out;
    }

    void mulCuda(
        FixedVector<float>& out, 
        FixedVector<float>& A, 
        FixedVector<float>& B
        )
    {
        assert(A.size() == B.size());
        size_t N = A.size();

        // 1.) Host pointers (CPU)
        const float*  hA   = A.data();
        const float*  hB   = B.data();
        float*  hOut = out.data();
        
        // 2.) Device Pointers (GPU)
        float* dA   = nullptr;  
        float* dB   = nullptr;
        float* dOut = nullptr;

        hipMallocAsync(&dA, N * sizeof(float),0);
        hipMallocAsync(&dB, N * sizeof(float),0);
        hipMallocAsync(&dOut, N * sizeof(float),0);

        // 3.) Copy input from host to device
        hipMemcpy(dA, hA, N * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(dB, hB, N * sizeof(float), hipMemcpyHostToDevice);

        // 4.) Launch Kernel
        int blockSize = 256; // This is mostly arbitrary! But we use 16 for a base 2 number less than 1024 that can still perform many operations.
        int gridSize = (N + blockSize - 1) / blockSize; 
        mulKernel<<<gridSize, blockSize>>>(dA, dB, dOut, N);
        // 5.) Copy the output from device to host
        // BEWARE: using vec.data() as a the T* works for all values EXCEPT BOOLS, as vectors of bools are NOT CONTIGUOUS.
        hipMemcpy(hOut,dOut, N * sizeof(float), hipMemcpyDeviceToHost);

        // 6.) Clean up
    }

    void mulCuda(
        FixedVector<FixedVector<float>>& out, 
        FixedVector<FixedVector<float>>& A, 
        FixedVector<FixedVector<float>>& B
        )
    {
        int m = A.size();  // Number of rows
        int n = A[0].size();  // Number of columns

        // Allocate device memory for matrices A, B, and out
        float* dA = nullptr;
        float* dB = nullptr;
        float* dOut = nullptr;

        // Allocate contiguous memory for all matrices (A, B, and out)
        hipMallocAsync((void**)&dA, m * n * sizeof(float), 0);
        checkCudaError("hipMalloc for dA");

        hipMallocAsync((void**)&dB, m * n * sizeof(float), 0);
        checkCudaError("hipMalloc for dB");

        hipMallocAsync((void**)&dOut, m * n * sizeof(float), 0);
        checkCudaError("hipMalloc for dOut");

        // Flatten the matrices and copy to device (single memory copy)
        float* hA = new float[m * n];
        float* hB = new float[m * n];


        // Flatten the 2D matrix A to 1D array
        for (int i = 0; i < m; ++i) {
            std::memcpy(hA + i * n, A[i].data(), n * sizeof(float));
            std::memcpy(hB + i * n, B[i].data(), n * sizeof(float));
        }
        
        // Copy the flattened matrices to device
        hipMemcpyAsync(dA, hA, m * n * sizeof(float), hipMemcpyHostToDevice);
        checkCudaError("hipMemcpy for dA");

        hipMemcpyAsync(dB, hB, m * n * sizeof(float), hipMemcpyHostToDevice);
        checkCudaError("hipMemcpy for dB");

        // Set up kernel launch parameters (using 16x16 block size)
        dim3 blockDim(16, 16);  // Block size: 16x16 threads
        dim3 gridDim((n + blockDim.x - 1) / blockDim.x, (m + blockDim.y - 1) / blockDim.y);  // Grid size

        // Launch the kernel for element-wise multiplication
        elementWiseMultiplyKernel<<<gridDim, blockDim>>>(dA, dB, dOut, m, n);
        checkCudaError("Kernel launch failed");

        // Copy the result back to the host in one go (flattened)
        float* hOut = new float[m * n];
        hipMemcpyAsync(hOut, dOut, m * n * sizeof(float), hipMemcpyDeviceToHost);
        checkCudaError("hipMemcpy for hOut");

        // Copy the flattened result back into the 2D out structure
        for (int i = 0; i < m; ++i) {
            std::memcpy(out[i].data(), hOut + i * n, n * sizeof(float));
        }

        // Free host memory
        delete[] hA;
        delete[] hB;
        delete[] hOut;
    }

    FixedVector<FixedVector<float>> linearCuda(
        FixedVector<FixedVector<float>> A, 
        FixedVector<FixedVector<float>> B,
        FixedVector<float> bias) 
        {

        int m = A.size();    // Number of rows in A
        int n = B[0].size(); // Number of columns in B
        int k = A[0].size(); // Number of columns in A (also the number of rows in B)

        
        // 1.) Flatten the 2D vectors into contiguous 1D arrays
        float* hA = new float[m * k];
        float* hB = new float[k * n];
        float* hBias = new float[n];
        float* hOut = new float[m * n];

        // Copy the values into the flattened arrays
        for (size_t i = 0; i < m; ++i) {
            memcpy(hA + i * k, A[i].data(), k * sizeof(float));  // Flatten row i of A
        }

        for (size_t i = 0; i < k; ++i) {
            memcpy(hB + i * n, B[i].data(), n * sizeof(float));  // Flatten row i of B
        }

        memcpy(hBias, bias.data(), n * sizeof(float));  // Copy the bias values
    

        // Allocate memory on the GPU only once
        static float* dA = nullptr;
        static float* dB = nullptr;
        static float* dBias = nullptr;
        static float* dOut = nullptr;
        static int prevM = -1, prevN = -1, prevK = -1;  // Track previous matrix dimensions
        // Check if the size has changed and reallocate if necessary
        if (prevM != m || prevK != k || prevN != n) {
            // If dimensions are different, free the previous memory and allocate new memory
            if (dA != nullptr) hipFree(dA);
            if (dB != nullptr) hipFree(dB);
            if (dOut != nullptr) hipFree(dOut);
            if (dBias != nullptr) hipFree(dBias);
            // Allocate memory for the new matrices on the device
            hipMallocAsync((void**)&dA, m * k * sizeof(float),0);
            checkCudaError("CudaMalloc for dA");
            hipMallocAsync((void**)&dB, k * n * sizeof(float),0);
            checkCudaError("CudaMalloc for dB");
            hipMallocAsync((void**)&dOut, m * n * sizeof(float),0);
            checkCudaError("CudaMalloc for dOut");
            hipMallocAsync((void**)&dBias, n * sizeof(float),0);
            checkCudaError("CudaMalloc for dBias");

            // Update the previous dimensions
            prevM = m;
            prevK = k;
            prevN = n;
        }


        // 2.) Copy data from host to device
        hipMemcpyAsync(dA, hA, m * k * sizeof(float), hipMemcpyHostToDevice);
        checkCudaError("hipMemcpyAsync for dA");
        hipMemcpyAsync(dB, hB, k * n * sizeof(float), hipMemcpyHostToDevice);
        checkCudaError("hipMemcpyAsync for dB");
        hipMemcpyAsync(dBias, hBias, n * sizeof(float), hipMemcpyHostToDevice);
        checkCudaError("hipMemcpyAsync for dBias");
        hipMemsetAsync(dOut, 0, m * n * sizeof(float),0);
        checkCudaError("hipMemsetAsync for dOut");

        // 3.) Launch Kernel
        dim3 blockDim(8, 8);  // Smaller block size to reduce resource usage
        dim3 gridDim((m + blockDim.x - 1) / blockDim.x, (n + blockDim.y - 1) / blockDim.y);  // Grid size based on matrix dimensions

        linear_kernel<<<gridDim, blockDim>>>(m, n, k, dA, dB, dBias, dOut);
        checkCudaError("Linear Kernel Failure");
        //start = std::chrono::high_resolution_clock::now();
        // 4.) Copy the result back to host
        hipMemcpyAsync(hOut, dOut, m * n * sizeof(float), hipMemcpyDeviceToHost,0);
        checkCudaError("hipMemcpyAsync for Hout");

        // 5.) Convert the result back to a FixedVector
        FixedVector<FixedVector<float>> result(m, FixedVector<float>(n));
        for (size_t i = 0; i < m; ++i) {
            memcpy(result[i].data(), hOut + i * n, n * sizeof(float));  // Copy each row to the result
        }

        // 6.) Clean up
        delete[] hA;
        delete[] hB;
        delete[] hBias;
        delete[] hOut;
        return result;
    }

    void addCuda(
        FixedVector<float>& A,
        FixedVector<float>& B
        )
    {
        // 1.) Host Pointers (CPU)
        float* hA = A.data();
        const float* hB = B.data();
        size_t N = A.size();

        // 2.) Allocate GPU memory
        float* dA = new float[N];
        float* dB = new float[N];

        hipMalloc(&dA, N * sizeof(float));
        hipMalloc(&dB, N * sizeof(float));

        // 3.) Copy input to GPU
        hipMemcpy(dA, hA, N * sizeof(float), hipMemcpyHostToDevice);
        checkCudaError("hipMalloc for dA");
        hipMemcpy(dB, hB, N * sizeof(float), hipMemcpyHostToDevice);
        checkCudaError("hipMalloc for dB");

        // 4.) Launch Kernel
        int blockSize = 256; // Still arbitrary...
        int gridSize = (N + blockSize - 1) / blockSize;
        addKernel<<<gridSize, blockSize>>>(dA, dB, N);
        checkCudaError("Kernel launch failed");

        // 5.) Copy device output to host
        hipMemcpy(hA, dA, N * sizeof(float), hipMemcpyDeviceToHost);
        checkCudaError("hipMemcpy for hOut");

        // 6.) Everybody clean up
        delete[] hA;
        delete[] hB;
        hipFree(dA);
        hipFree(dB);
    }


    void addCuda(
        FixedVector<FixedVector<float>>& A, 
        FixedVector<FixedVector<float>>& B
        ) {
        int m = A.size();     // Number of rows in A
        int n = A[0].size();  // Number of columns in A
        
        // Flatten the 2D vectors into contiguous 1D arrays
        float* hA = new float[m * n];
        float* hB = new float[m * n];

        for (size_t i = 0; i < m; ++i){
            memcpy(hA + i * n, A[i].data(), n * sizeof(float));
            memcpy(hB + i * n, B[i].data(), n * sizeof(float));
        }

        // Device memory allocations (only if dimensions have changed)
        static float* dA = nullptr;
        static float* dB = nullptr;
        static int prevM = -1, prevN = -1;

        // Check if the previous dimensions were the same, if not we need to 
        // free the pointers on the device
        if (prevM != m || prevN != n) {
            if (dA != nullptr) hipFree(dA);
            if (dB != nullptr) hipFree(dB);

            // Allocate device memory
            hipMallocAsync((void**)&dA, m * n * sizeof(float),0);
            checkCudaError("hipMalloc for dA");
            hipMallocAsync((void**)&dB, m * n * sizeof(float),0);
            checkCudaError("hipMalloc for dB");

            prevM = m;
            prevN = n;
        }

        // Copy A and B to device memory asynchronously
        hipMemcpyAsync(dA, hA, m * n * sizeof(float), hipMemcpyHostToDevice);
        checkCudaError("hipMemcpyAsync for dA");
        hipMemcpyAsync(dB, hB, m * n * sizeof(float), hipMemcpyHostToDevice);
        checkCudaError("hipMemcpyAsync for dB");

        // Launch the kernel
        dim3 blockDim(16, 16);
        dim3 gridDim((m + blockDim.x - 1) / blockDim.x, (n + blockDim.y - 1) / blockDim.y);  // Grid size based on matrix dimensions
        add2DKernel<<<gridDim, blockDim>>>(dA, dB, m, n);
        checkCudaError("Kernel launch failed");
        
        // Copy result back to host asynchronously
        hipMemcpyAsync(hA, dA, m * n * sizeof(float), hipMemcpyDeviceToHost);
        checkCudaError("hipMemcpyAsync for hOut");
        
        for (size_t i = 0; i < m; ++i) {
                memcpy(A[i].data(), hA + i * n, n * sizeof(float));  // Copy row i of the result
        }
        delete[] hA;
        delete[] hB;
    }
}