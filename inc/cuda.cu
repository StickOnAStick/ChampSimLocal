#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cassert>
#include "FixedVectorMath.hh"
#include <cmath>
#include <chrono>
#include <iostream>
#include <cudaProfiler.h>
#include <cstring>
#include <omp.h>

void checkCudaError(const char* message) {
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << message << " - " << hipGetErrorString(err) << std::endl;
        exit(-1);
    }
}
#define CEIL_DIV(M, N) (((M) + (N)-1) / (N))
#define TILE_SIZE 32
#define BLOCK_SIZE 32

void print_attention_scores_per_head(float* d_attention_scores, int num_heads, int sequence_len) {
    float* h_attention_scores = new float[num_heads * sequence_len * sequence_len];
    hipMemcpy(h_attention_scores, d_attention_scores, num_heads * sequence_len * sequence_len * sizeof(float), hipMemcpyDeviceToHost);

    for (int head = 0; head < num_heads; ++head) {
        std::cout << "Head " << head << " Attention Scores:\n";
        for (int i = 0; i < sequence_len; ++i) {
            for (int j = 0; j < sequence_len; ++j) {
                std::cout << h_attention_scores[head * sequence_len * sequence_len + i * sequence_len + j] << " ";
            }
            std::cout << "\n";
        }
        std::cout << "\n";
    }
    delete[] h_attention_scores;
}

void print_device_matrix(const float* d_matrix, int rows, int cols, const std::string& name, hipStream_t stream) {
    // Allocate host memory
    float* h_matrix = new float[rows * cols];

    // Copy from device to host
    hipMemcpyAsync(h_matrix, d_matrix, rows * cols * sizeof(float), hipMemcpyDeviceToHost,stream);
    hipDeviceSynchronize();  // Ensure memory copy is completed before printing

    // Print the first 5x5 submatrix
    std::cout << "First 5x5 submatrix of " << name << ":\n";
    for (int i = 0; i < 24 && i < rows; ++i) {  // Ensure within bounds
        std::cout << i << ":";
        for (int j = 0; j < 70 && j < cols; ++j) {
            std::cout << h_matrix[i * cols + j] << " ";
        }
        std::cout << "\n";
    }

    // Free host memory
    delete[] h_matrix;
}

void printmatrix(const FixedVector<FixedVector<float>>& matrix, const std::string& name) {
    std::cout << "Matrix: " << name << std::endl;
    for (size_t i = 0; i < std::min(matrix.size(), static_cast<size_t>(5)); ++i) {
        for (size_t j = 0; j < std::min(matrix[i].size(), static_cast<size_t>(5)); ++j) {
            std::cout << matrix[i][j] << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
  }
  
  void print_dmask(float* dmask, int m) {
    // Allocate host memory
    float* hmask = new float[m * m];

    // Copy from device to host
    hipMemcpy(hmask, dmask, m * m * sizeof(float), hipMemcpyDeviceToHost);

    // Print the first 5x5 submatrix
    std::cout << "First 5x5 submatrix of dmask:\n";
    for (int i = 0; i < 5 && i < m; ++i) {  // Ensure within bounds
        for (int j = 0; j < 5 && j < m; ++j) {
            std::cout << hmask[i * m + j] << " ";
        }
        std::cout << "\n";
    }

    // Free host memory
    delete[] hmask;
}

  void print_hmask(const float* hmask, int m) {
    std::cout << "First 5x5 submatrix of hmask:\n";
    for (int i = 0; i < 5 && i < m; ++i) {  // Ensure within bounds
        for (int j = 0; j < 5 && j < m; ++j) {
            std::cout << hmask[i * m + j] << " ";
        }
        std::cout << "\n";
    }
}
void printMatrix(const char* name, float* d_matrix, int rows, int cols) {
    FixedVector<float> h_matrix(rows * cols);
    hipMemcpy(h_matrix.data(), d_matrix, rows * cols * sizeof(float), hipMemcpyDeviceToHost);

    std::cout << name << " (First 5x5 values):\n";
    int display_rows = std::min(5, rows);
    int display_cols = std::min(5, cols);
    
    for (int i = 0; i < display_rows; ++i) {
        for (int j = 0; j < display_cols; ++j) {
            std::cout << h_matrix[i * cols + j] << " ";
        }
        std::cout << "\n";
    }
    std::cout << "--------------------------------------\n";
}
/*
    This file is split into two sections!

    1. Kernel code
        - This is the code executed on the GPU itself
    
    2. Cuda Definitions
        - Our API to interact with the GPU, responsible for data preperation, transmission, and return

    ////////////////////////////////////////
    // Important Variables
    ////////////////////////////////////////

    blockIdx: Which block (in a given dimension denoted by .(x,y,z)) the current thread belongs to.

    blockDim: How many threads per block along the given axis .(x, y, z)
        - Nvidia executes threads in groups of 32 called warps. It's optimal (not required) to keep block sizes a multiple of 32


    Refer to the Cheat Sheet for visualizing these dimensions. 
    https://www.eecs.umich.edu/courses/eecs471/resources/materials/CUDA-Thread-Indexing-Cheatsheet.pdf
*/

/*
-----------------------------------------
    Kernel function defintions
-----------------------------------------
*/


__global__ void mulKernel(const float* A, const float* B, float* out, size_t N){
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N){
        out[idx] = A[idx] * B[idx];
    }
}

__global__ void mul2DKernel(const float* A, const float* B, float* out, size_t width, size_t height){
    //1. Determine row and column for the current thread
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;

    // 2. Check Boundary 
    if (row < height && col < width){
        size_t idx = row * width + col;
        out[idx] = A[idx] * B[idx];
    }
}

__global__ void addKernel(float* A, const float* B, size_t N){
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N){
        A[idx] += B[idx];
    }
}

__global__ void add2DKernel(float* A, const float* B, size_t width, size_t height){
    //1. Determine row and column for the current thread
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;

    // 2. Check Boundary 
    if (row < height && col < width){
        size_t idx = row * width + col;
        A[idx] += B[idx];
    }
}

__global__ void linear_kernel(int M, int N, int K, const float *A, const float *B, const float *bias, float *C) {
    const uint x = blockIdx.x * blockDim.x + threadIdx.x;
    const uint y = blockIdx.y * blockDim.y + threadIdx.y;
    // keep within tile bounds 
    if (x < M && y < N) {
      float tmp = 0.0;
      for (int i = 0; i < K; ++i) 
        tmp += A[x * K + i] * B[i * N + y];
      C[x * N + y] =  tmp + C[x * N + y];
    }
    C[x * N + y] += bias[y];
}
// ---------------------------------
// 2) Example LN row kernel
// ---------------------------------
__global__ void layerNormRowKernel(
    const float* __restrict__ dIn,
    float* __restrict__ dOut,
    float* __restrict__ dMean,
    float* __restrict__ dVar,
    int rows,
    int cols,
    float epsilon
){
    int row = blockIdx.x; 
    if (row >= rows) return;

    // We'll do naive parallel sums with atomicAdd. 
    __shared__ float sMean;
    __shared__ float sVar;

    if (threadIdx.x == 0) {
        sMean = 0.0f;
        sVar  = 0.0f;
    }
    __syncthreads();

    int startIdx = row*cols;
    float localSum = 0.0f;
    for (int c = threadIdx.x; c < cols; c += blockDim.x) {
        localSum += dIn[startIdx + c];
    }
    atomicAdd(&sMean, localSum);
    __syncthreads();

    if (threadIdx.x == 0) {
        sMean /= (float)cols;
        dMean[row] = sMean;
    }
    __syncthreads();

    float localVar = 0.0f;
    for (int c = threadIdx.x; c < cols; c += blockDim.x) {
        float diff = dIn[startIdx + c] - sMean;
        localVar += diff*diff;
    }
    atomicAdd(&sVar, localVar);
    __syncthreads();

    if (threadIdx.x == 0) {
        sVar /= (float)cols;
        dVar[row] = sVar;
    }
    __syncthreads();

    float invStd = rsqrtf(sVar + epsilon);
    for (int c = threadIdx.x; c < cols; c += blockDim.x) {
        float val = dIn[startIdx + c];
        dOut[startIdx + c] = (val - sMean)*invStd;
    }
}

__global__ void dotKernel(
    const float* A, 
    const float* B, 
    float* C,
    int rowsA, 
    int colsA, 
    int colsB
) {
    // Compute row and column for this thread
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // If within valid range
    if (row < rowsA && col < colsB) {
        float sum = 0;
        // Multiply row of A by column of B
        for (int k = 0; k < colsA; ++k) {
            sum += A[row * colsA + k] * B[k * colsB + col];
        }
        C[row * colsB + col] = sum;
    }
}
#define TILE_SIZE 32
__global__ void sgemm_blockheads(int m, int n, int k, 
    const float* sequence_history, const float *w_q, const float *w_v, const float *w_k,
    float* Q, float* V, float* K) {

    __shared__ float tile_A[TILE_SIZE][TILE_SIZE];
    __shared__ float tile_Bq[TILE_SIZE][TILE_SIZE];
    __shared__ float tile_Bv[TILE_SIZE][TILE_SIZE];
    __shared__ float tile_Bk[TILE_SIZE][TILE_SIZE];

    int tx = threadIdx.x, ty = threadIdx.y;
    int row = blockIdx.y * TILE_SIZE + ty;
    int col = blockIdx.x * TILE_SIZE + tx;

    float sumQ = 0.0f, sumV = 0.0f, sumK = 0.0f;

    for (int t = 0; t < (k + TILE_SIZE - 1) / TILE_SIZE; ++t) {
        // Load elements into shared memory
        if (row < m && t * TILE_SIZE + tx < k) {
            tile_A[ty][tx] = sequence_history[row * k + t * TILE_SIZE + tx];
        } else {
            tile_A[ty][tx] = 0.0f;
        }

        if (t * TILE_SIZE + ty < k && col < n) {
            tile_Bq[ty][tx] = w_q[(t * TILE_SIZE + ty) * n + col];
            tile_Bv[ty][tx] = w_v[(t * TILE_SIZE + ty) * n + col];
            tile_Bk[ty][tx] = w_k[(t * TILE_SIZE + ty) * n + col];
        } else {
            tile_Bq[ty][tx] = 0.0f;
            tile_Bv[ty][tx] = 0.0f;
            tile_Bk[ty][tx] = 0.0f;
        }

        __syncthreads();

        // Compute partial sum within tile
        for (int i = 0; i < TILE_SIZE; ++i) {
            sumQ += tile_A[ty][i] * tile_Bq[i][tx];
            sumV += tile_A[ty][i] * tile_Bv[i][tx];
            sumK += tile_A[ty][i] * tile_Bk[i][tx];
        }
        __syncthreads();
    }

    if (row < m && col < n) {
        Q[row * n + col] += sumQ;
        V[row * n + col] += sumV;
        K[row * n + col] += sumK;
    }
}

__global__ void compute_attention_scores(
    const float* Q, const float* K, const float* mask,
    float* attention_scores, int sequence_len, int d_head,
    bool use_mask, int num_heads, int d_model) {

    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    int head = blockIdx.z;  // Each block in the z-dimension corresponds to a different head

    if (row >= sequence_len || col >= sequence_len || head >= num_heads) return;

    int head_offset = head * d_head;
    float score = 0.0f;

    // Compute raw attention scores (QK^T)
    for (int k = 0; k < d_head; ++k) {
        int Q_index = row * d_model + head_offset + k;
        int K_index = col * d_model + head_offset + k;
        score += Q[Q_index] * K[K_index];
    }

    // Scale the score
    score /= sqrtf(static_cast<float>(d_head));

    // Apply mask if necessary
    if (use_mask) {
        int mask_index = row * sequence_len + col;
        score += mask[mask_index];  // Ensure mask is correctly indexed
    }

    // Store attention score for this head
    int output_index = head * sequence_len * sequence_len + row * sequence_len + col;
    attention_scores[output_index] = score;
}


__global__ void compute_head_out(
    float* output, const float* att_score_soft, const float* V, 
    const int sequence_len, const int d_model, const int head, const int d_head) 
{
    const int row = blockIdx.x * blockDim.x + threadIdx.x;
    const int col = blockIdx.y * blockDim.y + threadIdx.y;
    
    int head_offset = head * d_head;

    if (row < sequence_len && col < d_head) {
        float tmp = 0.0f;
        for (int i = 0; i < sequence_len; ++i) {
            tmp += att_score_soft[row * sequence_len + i] * V[i * d_model + head_offset + col];
        }
        output[row * d_head + col] = tmp;
    }
}


__global__ void merge_heads(const float* head_out,  float* attention_out, int sequence_len, int d_model, int num_heads, int d_head, int head) // Head index passed as input
{
    int seq_idx = blockIdx.x * blockDim.x + threadIdx.x; // Row index (sequence)
    int feature_idx = blockIdx.y * blockDim.y + threadIdx.y; // Column index (within d_head)

    if (seq_idx < sequence_len && feature_idx < d_head) {
        // Compute correct input index in head_out for this head
        int head_out_index = seq_idx * d_head + feature_idx;

        // Compute correct output index in attention_out
        int attention_out_index = seq_idx * d_model + head * d_head + feature_idx;
        __syncthreads();
        attention_out[attention_out_index] = head_out[head_out_index];
    }
}


__global__ void softMax(float* output, float* input, int M, int N) 
{
    int row = blockDim.x * blockIdx.x + threadIdx.x;

    if (row < M) {
        // maximum of this row
        float x_max = -INFINITY;
        // norm factor of this row
        float norm = 0.0f;

        // output in 3 passes
        for (int col = 0; col < N; col++) {
            int i = row * N + col;
            x_max = max(x_max, input[i]);
        }
        for (int col = 0; col < N; col++) {
            int i = row * N + col;
            norm += expf(input[i] - x_max);
        }
        for (int col = 0; col < N; col++) {
            int i = row * N + col;
            output[i] = expf(input[i] - x_max) / norm;
        }
    }
}

#define BLOCK_SIZE 32
/*
-----------------------------------------
    Cuda API Definitions
-----------------------------------------
*/   
namespace FixedVectorMath {

    FixedVector<FixedVector<float>> MMA_CUDA(
        bool use_mask,
        int sequence_len,
        int d_model,
        const int num_ma_heads, 
        FixedVector<FixedVector<float>> &sequence_history,
        FixedVector<FixedVector<float>> &w_q,
        FixedVector<FixedVector<float>> &w_k,
        FixedVector<FixedVector<float>> &w_v,
        FixedVector<FixedVector<float>> &w_o) {
         
        hipStream_t streams[num_ma_heads];
        for (int i = 0; i < num_ma_heads; i++)
            hipStreamCreate(&streams[i]);
    
        FixedVector<FixedVector<float>> attention_out(sequence_len, FixedVector<float>(d_model, 0.0f));
        FixedVector<FixedVector<float>> mask(sequence_len, FixedVector<float>(sequence_len, 0.0f));
       
        if (use_mask)
          FixedVectorMath::applyMask(mask);
        

        // Fix dimensions
        int m = sequence_len;  // Number of sequences
        int n = d_model;       // Model dimension
        int k = d_model;       // Embedding dimension
        int d_head = d_model / num_ma_heads;
        int num_heads = num_ma_heads;

        // Allocate host memory
        float* hsequence_history = new float[m * k];
        float* hmask = new float[m * m];  // Flattened mask
        float* hout = new float[m * d_model];  
        float* hw_q = new float[k * d_model];
        float* hw_v = new float[k * d_model];
        float* hw_k = new float[k * d_model];
        
        for (size_t i = 0; i < m; ++i) {
            for (size_t j = 0; j < m; ++j) {
                hmask[i * m + j] = mask[i][j];  // Flattening row-major order
            }
        }
       
        // Copy input matrices to 1D arrays
        for (size_t i = 0; i < m; ++i) {
            memcpy(hsequence_history + i * k, sequence_history[i].data(), k * sizeof(float));
        }
        for (size_t i = 0; i < k; ++i) {
            memcpy(hw_q + i * d_model, w_q[i].data(), d_model * sizeof(float));
            memcpy(hw_v + i * d_model, w_v[i].data(), d_model * sizeof(float));
            memcpy(hw_k + i * d_model, w_k[i].data(), d_model * sizeof(float));
        }
    
       // Device memory
        static float* dsequence_history = nullptr;
        static float* dattention_scores = nullptr;
        static float* dattention_scores_softmax = nullptr;
        static float* dOut = nullptr;
        static float* dw_q = nullptr;
        static float* dmask = nullptr;
        static float* dw_k = nullptr;
        static float* dw_v = nullptr;
        static float* dQ = nullptr;
        static float* dV = nullptr;
        static float* dK = nullptr;
        static float* dhead_out = nullptr;
        static float* dattention_out = nullptr;

        if (dsequence_history == nullptr) {
            // Free previously allocated memory (if any)
            hipFree(dw_q);
            hipFree(dw_k);
            hipFree(dw_v);
            hipFree(dmask);
            hipFree(dsequence_history);
            hipFree(dattention_scores);
            hipFree(dQ);
            hipFree(dV);
            hipFree(dOut);
            hipFree(dK);
            hipFree(dhead_out);
            hipFree(dattention_out);

            // Allocate device memory
            hipMalloc((void**)&dsequence_history, m * k * sizeof(float));
            checkCudaError("hipMalloc for dsequence_history");
            
            hipMalloc((void**)&dw_q, k * d_model * sizeof(float));
            checkCudaError("hipMalloc for dw_q");
            
            hipMalloc((void**)&dw_k, k * d_model * sizeof(float));
            checkCudaError("hipMalloc for dw_k");
            
            hipMalloc((void**)&dw_v, k * d_model * sizeof(float));
            checkCudaError("hipMalloc for dw_v");
            
            hipMalloc((void**)&dQ, sequence_len * d_model * sizeof(float));
            checkCudaError("hipMalloc for dQ");
            
            hipMalloc((void**)&dV, sequence_len * d_model * sizeof(float));
            checkCudaError("hipMalloc for dV");
            
            hipMalloc((void**)&dK, sequence_len * d_model * sizeof(float));
            checkCudaError("hipMalloc for dK");
            
            hipMalloc((void**)&dOut, sequence_len * d_model * sizeof(float));
            checkCudaError("hipMalloc for dOut");
            
            hipMalloc((void**)&dattention_scores, num_ma_heads*m * m * sizeof(float));
            checkCudaError("hipMalloc for dattention_scores");

            hipMalloc((void**)&dattention_scores_softmax, m * m * sizeof(float));
            checkCudaError("hipMalloc for dattention_scores");
            
            hipMalloc((void**)&dmask, m * m * sizeof(float));
            checkCudaError("hipMalloc for dmask");
            
            // Allocate memory for head_out (stores attention-weighted V output)
            hipMalloc((void**)&dhead_out, sequence_len * d_head * sizeof(float));
            checkCudaError("hipMalloc for dhead_out");

            // Allocate memory for final attention_out (concatenated head_out results)
            hipMalloc((void**)&dattention_out, sequence_len * d_model * sizeof(float));
            checkCudaError("hipMalloc for dattention_out");
        }

    // Copy host data to device memory
    hipMemcpyAsync(dmask, hmask, m * m * sizeof(float), hipMemcpyHostToDevice,0);
    hipMemcpyAsync(dsequence_history, hsequence_history, m * k * sizeof(float), hipMemcpyHostToDevice,0);
    hipMemcpyAsync(dw_q, hw_q, k * d_model * sizeof(float), hipMemcpyHostToDevice,0);
    hipMemcpyAsync(dw_v, hw_v, k * d_model * sizeof(float), hipMemcpyHostToDevice,0);
    hipMemcpyAsync(dw_k, hw_k, k * d_model * sizeof(float), hipMemcpyHostToDevice,0);

    // Initialize device output matrices
    hipMemsetAsync(dQ, 0, sequence_len * d_model * sizeof(float),0);
    hipMemsetAsync(dV, 0, sequence_len * d_model * sizeof(float),0);
    hipMemsetAsync(dK, 0, sequence_len * d_model * sizeof(float),0);
    hipMemsetAsync(dattention_scores, 0, num_ma_heads*m * m * sizeof(float),0);
    hipMemsetAsync(dattention_scores_softmax, 0, m * m * sizeof(float),0);
    hipMemsetAsync(dhead_out, 0, sequence_len * d_head * sizeof(float),0);
    hipMemsetAsync(dattention_out, 0, sequence_len * d_model * sizeof(float), 0);

    // Kernel configuration
    dim3 blockDim1(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim1((n + BLOCK_SIZE - 1) / BLOCK_SIZE, (m + BLOCK_SIZE - 1) / BLOCK_SIZE);        
    sgemm_blockheads<<<gridDim1, blockDim1, 0, 0>>>(m, n, k, dsequence_history, dw_q, dw_v, dw_k, dQ, dV, dK);
    hipDeviceSynchronize();

    dim3 blockSize(8, 8, 1);  // Adjust based on available resources
    dim3 gridSize(
    (sequence_len + blockSize.x - 1) / blockSize.x,
    (sequence_len + blockSize.y - 1) / blockSize.y,
     num_heads);
    // Each head gets its own block in the z-dimension

    compute_attention_scores<<<gridSize, blockSize>>>(dQ, dK, dmask, dattention_scores, sequence_len, d_head, use_mask, num_ma_heads, d_model);
    print_attention_scores_per_head(dattention_scores, num_heads, sequence_len);
    hipDeviceSynchronize();
    // dim3 blockDim1(16, 16);
    // dim3 gridDim1((d_model + blockDim.x - 1) / blockDim.x, (d_model + blockDim.y - 1) / blockDim.y); 
    // for(int head = 0; head < num_ma_heads; ++head)
    // {
    //     compute_attention_scores<<<gridDim1, blockDim1,0,streams[head]>>>(dQ, dK, dmask, dattention_scores, sequence_len, d_head, use_mask, head, d_model);
    //     softMax<<<gridDim1, blockDim1,0,streams[head]>>>(dattention_scores_softmax,dattention_scores,sequence_len,sequence_len);
    //     compute_head_out<<<gridDim1, blockDim1,0,streams[head]>>>(dhead_out, dattention_scores_softmax,dV, sequence_len,d_model,head, d_head);
    //     hipDeviceSynchronize();
    //     merge_heads<<<gridDim1, blockDim1,0,streams[head]>>>(dhead_out, dattention_out, sequence_len,d_model, num_ma_heads, d_head,head);
    //     // print_device_matrix(dattention_scores, sequence_len, sequence_len, "dattention_out",streams[0]);
    //     // print_device_matrix(dhead_out, sequence_len, d_head, "dhead_out",streams[0]);
    // }
    // print_device_matrix(dattention_out, sequence_len, d_model, "dattention_out",streams[0]);

        // Cleanup
        delete[] hsequence_history;
        delete[] hout;
        delete[] hmask;
        delete[] hw_q;
        delete[] hw_v;
        delete[] hw_k;

        for (int i = 0; i < num_ma_heads; i++)
            hipStreamDestroy(streams[i]);
        
        return attention_out;
    }
    
    void normalizeCuda(
        FixedVector<FixedVector<float>>& matrix,
        FixedVector<float>& means,
        FixedVector<float>& vars,
        float epsilon
    ) {
        int m = matrix.size();
        int n = matrix[0].size();
    
        if ((int)means.size() != m) {
            means = FixedVector<float>(m, 0.0f);
        }
        if ((int)vars.size() != m) {
            vars = FixedVector<float>(m, 0.0f);
        }
    
        // flatten to host
        float* hIn = new float[m*n];
        for (int i = 0; i < m; i++) {
            memcpy(hIn + i*n, matrix[i].data(), n*sizeof(float));
        }
    
        // allocate GPU
        float* dIn=nullptr; 
        float* dOut=nullptr; 
        float* dMean=nullptr; 
        float* dVar=nullptr;
        hipMalloc(&dIn,  m*n*sizeof(float));
        hipMalloc(&dOut, m*n*sizeof(float));
        hipMalloc(&dMean,m*sizeof(float));
        hipMalloc(&dVar, m*sizeof(float));
    
        // copy input
        hipMemcpy(dIn, hIn, m*n*sizeof(float), hipMemcpyHostToDevice);
    
        // launch kernel: 1 block per row, up to 256 threads. 
        dim3 grid(m);
        dim3 block(256);
        layerNormRowKernel<<<grid, block>>>(dIn, dOut, dMean, dVar, m, n, epsilon);
        checkCudaError("layerNormRowKernel");
    
        // copy back
        hipMemcpy(hIn,   dOut,  m*n*sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(means.data(), dMean, m*sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(vars.data(),  dVar,  m*sizeof(float), hipMemcpyDeviceToHost);
    
        // store result
        for (int i = 0; i < m; i++) {
            memcpy(matrix[i].data(), hIn + i*n, n*sizeof(float));
        }
    
        // free
        hipFree(dIn);
        hipFree(dOut);
        hipFree(dMean);
        hipFree(dVar);
        delete[] hIn;
    }

    FixedVector<FixedVector<float>> dotCuda(
        const FixedVector<FixedVector<float>>& A,
        const FixedVector<FixedVector<float>>& B
    ){
    // Ensure A's columns == B's rows
    int rowsA = A.size();
    int colsA = A[0].size();
    int rowsB = B.size();
    int colsB = B[0].size();

    if (colsA != rowsB) {
        throw std::invalid_argument("Matrix dimension mismatch: A[rows x cols], B[cols x ???]");
    }

    // Flatten A and B for memory transfer to device
    std::vector<float> h_A(rowsA * colsA);
    std::vector<float> h_B(rowsB * colsB);
    for (int i = 0; i < rowsA; ++i) {
        for (int j = 0; j < colsA; ++j) {
            h_A[i * colsA + j] = A[i][j];
        }
    }
    for (int i = 0; i < rowsB; ++i) {
        for (int j = 0; j < colsB; ++j) {
            h_B[i * colsB + j] = B[i][j];
        }
    }

    // Allocate memory on the device
    float* d_A = nullptr;
    float* d_B = nullptr;
    float* d_C = nullptr;
    size_t sizeA = rowsA * colsA * sizeof(float);
    size_t sizeB = rowsB * colsB * sizeof(float);
    size_t sizeC = rowsA * colsB * sizeof(float);

    hipMalloc((void**)&d_A, sizeA);
    hipMalloc((void**)&d_B, sizeB);
    hipMalloc((void**)&d_C, sizeC);

    // Copy data from host to device
    hipMemcpy(d_A, h_A.data(), sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B.data(), sizeB, hipMemcpyHostToDevice);

    // Choose reasonable block and grid sizes
    // For simplicity, we set the block to 16x16 threads
    dim3 block(16, 16);
    // We then compute how many blocks we need in each dimension
    dim3 grid((colsB + block.x - 1) / block.x,
              (rowsA + block.y - 1) / block.y);

    // Launch the kernel
    dotKernel<<<grid, block>>>(d_A, d_B, d_C, rowsA, colsA, colsB);

    // Wait for GPU to finish
    hipDeviceSynchronize();

    // Copy result back to host
    std::vector<float> h_C(rowsA * colsB);
    hipMemcpy(h_C.data(), d_C, sizeC, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Reshape flattened C back into 2D vector
    FixedVector<FixedVector<float>> result(rowsA, FixedVector<float>(colsB, static_cast<float>(0)));
    for (int i = 0; i < rowsA; ++i) {
        for (int j = 0; j < colsB; ++j) {
            result[i][j] = h_C[i * colsB + j];
        }
    }

    return result;
    }

    FixedVector<FixedVector<float>> linearCuda(
        FixedVector<FixedVector<float>> A, 
        FixedVector<FixedVector<float>> B,
        FixedVector<float> bias) 
        {

        int m = A.size();    // Number of rows in A
        int n = B[0].size(); // Number of columns in B
        int k = A[0].size(); // Number of columns in A (also the number of rows in B)

        
        // 1.) Flatten the 2D vectors into contiguous 1D arrays
        float* hA = new float[m * k];
        float* hB = new float[k * n];
        float* hBias = new float[n];
        float* hOut = new float[m * n];

        // Copy the values into the flattened arrays
        for (size_t i = 0; i < m; ++i) {
            memcpy(hA + i * k, A[i].data(), k * sizeof(float));  // Flatten row i of A
        }

        for (size_t i = 0; i < k; ++i) {
            memcpy(hB + i * n, B[i].data(), n * sizeof(float));  // Flatten row i of B
        }

        memcpy(hBias, bias.data(), n * sizeof(float));  // Copy the bias values
    

        // Allocate memory on the GPU only once
        static float* dA = nullptr;
        static float* dB = nullptr;
        static float* dBias = nullptr;
        static float* dOut = nullptr;
        static int prevM = -1, prevN = -1, prevK = -1;  // Track previous matrix dimensions
        // Check if the size has changed and reallocate if necessary
        if (prevM != m || prevK != k || prevN != n) {
            // If dimensions are different, free the previous memory and allocate new memory
            if (dA != nullptr) hipFree(dA);
            if (dB != nullptr) hipFree(dB);
            if (dOut != nullptr) hipFree(dOut);
            if (dBias != nullptr) hipFree(dBias);
            // Allocate memory for the new matrices on the device
            hipMallocAsync((void**)&dA, m * k * sizeof(float), 0);
            checkCudaError("CudaMalloc for dA");
            hipMallocAsync((void**)&dB, k * n * sizeof(float), 0);
            checkCudaError("CudaMalloc for dB");
            hipMallocAsync((void**)&dOut, m * n * sizeof(float), 0);
            checkCudaError("CudaMalloc for dOut");
            hipMallocAsync((void**)&dBias, n * sizeof(float), 0);
            checkCudaError("CudaMalloc for dBias");

            // Update the previous dimensions
            prevM = m;
            prevK = k;
            prevN = n;
        }


        // 2.) Copy data from host to device
        hipMemcpyAsync(dA, hA, m * k * sizeof(float), hipMemcpyHostToDevice);
        checkCudaError("hipMemcpyAsync for dA");
        hipMemcpyAsync(dB, hB, k * n * sizeof(float), hipMemcpyHostToDevice);
        checkCudaError("hipMemcpyAsync for dB");
        hipMemcpyAsync(dBias, hBias, n * sizeof(float), hipMemcpyHostToDevice);
        checkCudaError("hipMemcpyAsync for dBias");
        hipMemsetAsync(dOut, 0, m * n * sizeof(float),0);
        checkCudaError("hipMemsetAsync for dOut");

        // 3.) Launch Kernel
        dim3 blockDim(8, 8);  // Smaller block size to reduce resource usage
        dim3 gridDim((m + blockDim.x - 1) / blockDim.x, (n + blockDim.y - 1) / blockDim.y);  // Grid size based on matrix dimensions

        linear_kernel<<<gridDim, blockDim, m*n*sizeof(float), 0>>>(m, n, k, dA, dB, dBias, dOut);
        checkCudaError("Linear Kernel Failure");

        //start = std::chrono::high_resolution_clock::now();
        // 4.) Copy the result back to host
        hipStreamSynchronize(0); // Sync default stram before using dOut on another stream
        hipMemcpyAsync(hOut, dOut, m * n * sizeof(float), hipMemcpyDeviceToHost, 0);
        checkCudaError("hipMemcpyAsync for Hout");

        // 5.) Convert the result back to a FixedVector
        FixedVector<FixedVector<float>> result(m, FixedVector<float>(n));
        for (size_t i = 0; i < m; ++i) {
            memcpy(result[i].data(), hOut + i * n, n * sizeof(float));  // Copy each row to the result
        }

        // 6.) Clean up
        delete[] hA;
        delete[] hB;
        delete[] hBias;
        delete[] hOut;
        return result;
    }

    void addCuda(
        FixedVector<float>& A,
        FixedVector<float>& B
        )
    {
        // 1.) Host Pointers (CPU)
        float* hA = A.data();
        const float* hB = B.data();
        size_t N = A.size();

        // 2.) Allocate GPU memory
        float* dA;
        float* dB;

        hipMalloc(&dA, N * sizeof(float));
        hipMalloc(&dB, N * sizeof(float));

        // 3.) Copy input to GPU
        hipMemcpy(dA, hA, N * sizeof(float), hipMemcpyHostToDevice);
        checkCudaError("hipMalloc for dA");
        hipMemcpy(dB, hB, N * sizeof(float), hipMemcpyHostToDevice);
        checkCudaError("hipMalloc for dB");

        // 4.) Launch Kernel
        int blockSize = 256; // Still arbitrary...
        int gridSize = (N + blockSize - 1) / blockSize;
        addKernel<<<gridSize, blockSize>>>(dA, dB, N);
        checkCudaError("Kernel launch failed");

        // 5.) Copy device output to host
        hipMemcpy(hA, dA, N * sizeof(float), hipMemcpyDeviceToHost);
        checkCudaError("hipMemcpy for hOut");

        // 6.) Everybody clean up
        delete[] hA;
        delete[] hB;
        hipFree(dA);
        hipFree(dB);
    }

    void addCuda(
        FixedVector<FixedVector<float>>& A, 
        FixedVector<FixedVector<float>>& B
        ) {
        int m = A.size();     // Number of rows in A
        int n = A[0].size();  // Number of columns in A
        
        // Flatten the 2D vectors into contiguous 1D arrays
        float* hA = new float[m * n];
        float* hB = new float[m * n];

        for (size_t i = 0; i < m; ++i){
            memcpy(hA + i * n, A[i].data(), n * sizeof(float));
            memcpy(hB + i * n, B[i].data(), n * sizeof(float));
        }

        // Device memory allocations (only if dimensions have changed)
        static float* dA = nullptr;
        static float* dB = nullptr;
        static int prevM = -1, prevN = -1;

        // Check if the previous dimensions were the same, if not we need to 
        // free the pointers on the device
        if (prevM != m || prevN != n) {
            if (dA != nullptr) hipFree(dA);
            if (dB != nullptr) hipFree(dB);

            // Allocate device memory
            hipMallocAsync((void**)&dA, m * n * sizeof(float),0);
            checkCudaError("hipMalloc for dA");
            hipMallocAsync((void**)&dB, m * n * sizeof(float),0);
            checkCudaError("hipMalloc for dB");

            prevM = m;
            prevN = n;
        }

        // Copy A and B to device memory asynchronously
        hipMemcpyAsync(dA, hA, m * n * sizeof(float), hipMemcpyHostToDevice);
        checkCudaError("hipMemcpyAsync for dA");
        hipMemcpyAsync(dB, hB, m * n * sizeof(float), hipMemcpyHostToDevice);
        checkCudaError("hipMemcpyAsync for dB");

        // Launch the kernel
        dim3 blockDim(16, 16);
        dim3 gridDim((m + blockDim.x - 1) / blockDim.x, (n + blockDim.y - 1) / blockDim.y);  // Grid size based on matrix dimensions
        add2DKernel<<<gridDim, blockDim>>>(dA, dB, m, n);
        checkCudaError("Kernel launch failed");
        
        // Copy result back to host asynchronously
        hipMemcpyAsync(hA, dA, m * n * sizeof(float), hipMemcpyDeviceToHost);
        checkCudaError("hipMemcpyAsync for hOut");
        
        for (size_t i = 0; i < m; ++i) {
                memcpy(A[i].data(), hA + i * n, n * sizeof(float));  // Copy row i of the result
        }
        delete[] hA;
        delete[] hB;
    }
}