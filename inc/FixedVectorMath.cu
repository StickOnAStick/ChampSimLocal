#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cassert>
#include "FixedVectorMath.hh"


/*
    This file is split into two sections!

    1. Kernel code
        - This is the code executed on the GPU itself
    2. Cuda Definitions
        - Our API to interact with the GPU, responsible for data preperation, transmission, and return

    ////////////////////////////////////////
    // Important Variables
    ////////////////////////////////////////

    blockIdx: Which block (in a given dimension denoted by .(x,y,z)) the current thread belongs to.

    blockDim: How many threads per block along the given axis .(x, y, z)
        - Nvidia executes threads in groups of 32 called warps. It's optimal (not required) to keep block sizes a multiple of 32


    Refer to the Cheat Sheet for visualizing these dimensions. 
    https://www.eecs.umich.edu/courses/eecs471/resources/materials/CUDA-Thread-Indexing-Cheatsheet.pdf
*/

/*
-----------------------------------------
    Kernel function defintions
-----------------------------------------
*/
template <typename T>
__global__ void mulKernel(const T* A, const T* B, T* out, size_t N){
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N){
        out[idx] = A[idx] * B[idx];
    }
}

template <typename T>
__global__ void mul2DKernel(const T* A, const T* B, T* out, size_t width, size_t height){
    //1. Determine row and column for the current thread
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;

    // 2. Check Boundary 
    if (row < height && col < width){
        size_t idx = row * width + col;

        out[idx] = A[idx] * B[idx];
    }
}

template <typename T>
__global__ void addKernel(const T* A, const T* B, T* out, size_t N){
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N){
        out[idx] = A[idx] + B[idx];
    }
}

template <typename T>
__global__ void add2DKernel(const T* A, const T* B, T* out, size_t width, size_t height){
    //1. Determine row and column for the current thread
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;

    // 2. Check Boundary 
    if (row < height && col < width){
        size_t idx = row * width + col;

        out[idx] = A[idx] + B[idx];
    }
}

/*
-----------------------------------------
    Cuda API Definitions
-----------------------------------------
*/
template <typename T>
void mulCuda(FixedVector<T>& out, const FixedVector<T>& A, const FixedVector<T>& B){
    assert(A.size() == B.size());
    assert(out.size() == A.size());
    size_t N = A.size();

    // 1.) Host pointers (CPU)
    const T* hA   = A.data();
    const T* hB   = B.data();
    T*       hOut = out.data();

    // 2.) Device Pointers (GPU)
    T* dA   = nullptr;  
    T* dB   = nullptr;
    T* dOut = nullptr;

    hipMalloc(&dA, N * sizeof(T));
    hipMalloc(&dB, N * sizeof(T));
    hipMalloc(&dOut, N * sizeof(T));

    // 3.) Copy input from host to device
    hipMemcpy(dA, hA, N * sizeof(T), hipMemcpyHostToDevice);
    hipMemcpy(dB, hB, N * sizeof(T), hipMemcpyHostToDevice);

    // 4.) Launch Kernel
    int blockSize = 16; // This is mostly arbitrary! But we use 16 for a base 2 number less than 1024 that can still perform many operations.
    int gridSize = (N + blockSize - 1) / blockSize; 
    mulKernel<T><<<gridSize, blockSize>>>(dA, dB, dOut, N);

    // 5.) Copy the output from device to host
    hipMemcpy(hOut, dOut, N*sizeof(T), hipMemcpyDeviceToHost);

    // 6.) Clean up
    hipFree(dA);
    hipFree(dB);
    hipFree(dOut);

}